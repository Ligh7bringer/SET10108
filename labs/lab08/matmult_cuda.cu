#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>


using namespace std;

constexpr unsigned int ELEMENTS = 32;

// cuda kernel
__global__ void simple_multiply(float *output_C, unsigned int width_A, unsigned int height_A, unsigned int width_B, unsigned int height_B, const float *input_A, const float *input_B)
{
	// Get global position in Y direction
	unsigned int row = (blockIdx.y * 1024) + threadIdx.y;
	// Get global position in X direction
	unsigned int col = (blockIdx.x * 1024) + threadIdx.x;

	float sum = 0.0f;

	// Calculate result of one element of matrix C
	for (unsigned int i = 0; i < width_A; ++i)
		sum += input_A[row * width_A + i] * input_B[i * width_B + col];

	// Store result in matrix C
	output_C[row * width_B + col] = sum;
}

int main(int argc, char** argv) {
	// create host memory
	auto data_size = sizeof(float) * ELEMENTS * ELEMENTS;
	vector<vector<float>> A(ELEMENTS, vector<float>(ELEMENTS, 1));    // input aray, fill with ones
	vector<vector<float>> B(ELEMENTS, vector<float>(ELEMENTS, 1));    // input array, fill with ones
	vector<vector<float>> C(ELEMENTS, vector<float>(ELEMENTS));    // output array, fill with zeroes

	// create buffers
	float *buf_A, *buf_B, *buf_C;

	// init buffers
	hipMalloc((void**)&buf_A, data_size);
	hipMalloc((void**)&buf_B, data_size);
	hipMalloc((void**)&buf_C, data_size);

	// send data to gpu
	hipMemcpy(buf_A, &A[0], data_size, hipMemcpyHostToDevice);
	hipMemcpy(buf_B, &B[0], data_size, hipMemcpyHostToDevice);

	dim3 blocksPerGrid(512, 512);
	dim3 threadsPerBlock(32, 32);	
	// run kernel
	simple_multiply<<<blocksPerGrid, threadsPerBlock>>>(buf_C, ELEMENTS, ELEMENTS, ELEMENTS, ELEMENTS, buf_A, buf_B);

	// wait for kernel to complete
	hipDeviceSynchronize();

	// check for errors
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(error));
		return -1;
	}

	// read output buffer back to host
	hipMemcpy(&C[0], buf_C, data_size, hipMemcpyDeviceToHost);
	   
	// clean up
	hipFree(buf_A);
	hipFree(buf_B);
	hipFree(buf_C);

	// check results
	for (unsigned int row = 0; row < ELEMENTS; ++row) {
		for (unsigned int col = 0; col < ELEMENTS; ++col) {
			//cout << C[row][col] << ", ";
		}
		cout << endl;
	}

	return 0;
}