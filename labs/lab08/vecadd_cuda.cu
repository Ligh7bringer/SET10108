#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>


using namespace std;

constexpr size_t ELEMENTS = 2048;

__global__ void vecadd(const int *A, const int *B, int *C) {
	// get block index
	unsigned int block_idx = blockIdx.x;
	// get thread index
	unsigned int thread_idx = threadIdx.x;
	// get number of threads per block
	unsigned int block_dim = blockDim.x;
	// get thread unique ID
	unsigned int idx = (block_idx * block_dim) + thread_idx;
	// add A and B and store in C
	C[idx] = A[idx] + B[idx];
}

int main(int argc, char** argv) {
	// create host memory
	auto data_size = sizeof(int) * ELEMENTS;
	vector<int> A(ELEMENTS);    // input aray
	vector<int> B(ELEMENTS);    // input array
	vector<int> C(ELEMENTS);    // output array

	// initialise input data
	for (unsigned int i = 0; i < ELEMENTS; ++i)
		A[i] = B[i] = i;

	// declare buffers
	int *buffer_A, *buffer_B, *buffer_C;

	// init buffers
	hipMalloc((void**)&buffer_A, data_size);
	hipMalloc((void**)&buffer_B, data_size);
	hipMalloc((void**)&buffer_C, data_size);
	
	// write host data to device
	hipMemcpy(buffer_A, &A[0], data_size, hipMemcpyHostToDevice);
	hipMemcpy(buffer_B, &B[0], data_size, hipMemcpyHostToDevice);

	// run kernel with one thread for each element
	vecadd<<<ELEMENTS / 1024, 1024>>>(buffer_A, buffer_B, buffer_C);

	// wait for kernel to complete
	hipDeviceSynchronize();

	// read output buffer back to host
	hipMemcpy(&C[0], buffer_C, data_size, hipMemcpyDeviceToHost);

	// clean up
	hipFree(buffer_A);
	hipFree(buffer_B);
	hipFree(buffer_C);

	// Test that the results are correct
	for (int i = 0; i < 2048; ++i)
		if (C[i] != i + i) {
			cout << "Error: " << i << endl;
			break;
		} else
			cout << C[i] << endl;

	cout << "Finished" << endl;

	return 0;
}